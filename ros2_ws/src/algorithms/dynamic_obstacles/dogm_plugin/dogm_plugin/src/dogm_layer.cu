#include "hip/hip_runtime.h"
#include "dogm_plugin/dogm_layer.h"

#include <opencv2/opencv.hpp>
#include <opencv2/core/cuda.hpp>
#include <opencv2/cudawarping.hpp>

#include <iostream>

namespace dogm_plugin {

__global__ void setUnknownAsFree(cv::cuda::PtrStepSzi occupancy_grid);
__global__ void fillMeasurementGrid(dogm::MeasurementCell* __restrict__ measurement_grid, const cv::cuda::PtrStepSzi source,
                                    float occupancy_threshold);

DogmLayer::DogmLayer() {}

void DogmLayer::onInitialize() {
    declareParameter("enabled", rclcpp::ParameterValue(true));
    node_->get_parameter(name_ + "." + "enabled", enabled_);
    declareParameter("motion_compensation", rclcpp::ParameterValue(true));
    node_->get_parameter(name_ + "." + "motion_compensation", motion_compensation_);
    declareParameter("size", rclcpp::ParameterValue(50.0f));
    node_->get_parameter(name_ + "." + "size", params_.size);
    declareParameter("resolution", rclcpp::ParameterValue(0.2f));
    node_->get_parameter(name_ + "." + "resolution", params_.resolution);
    declareParameter("particle_count", rclcpp::ParameterValue(3 * static_cast<int>(1e6)));
    node_->get_parameter(name_ + "." + "particle_count", params_.particle_count);
    declareParameter("new_born_particle_count", rclcpp::ParameterValue(3 * static_cast<int>(1e5)));
    node_->get_parameter(name_ + "." + "new_born_particle_count", params_.new_born_particle_count);
    declareParameter("persistence_prob", rclcpp::ParameterValue(0.99f));
    node_->get_parameter(name_ + "." + "persistence_prob", params_.persistence_prob);
    declareParameter("stddev_process_noise_position", rclcpp::ParameterValue(0.1f));
    node_->get_parameter(name_ + "." + "stddev_process_noise_position", params_.stddev_process_noise_position);
    declareParameter("stddev_process_noise_velocity", rclcpp::ParameterValue(1.0f));
    node_->get_parameter(name_ + "." + "stddev_process_noise_velocity", params_.stddev_process_noise_velocity);
    declareParameter("birth_prob", rclcpp::ParameterValue(0.02f));
    node_->get_parameter(name_ + "." + "birth_prob", params_.birth_prob);
    declareParameter("stddev_velocity", rclcpp::ParameterValue(30.0f));
    node_->get_parameter(name_ + "." + "stddev_velocity", params_.stddev_velocity);
    declareParameter("init_max_velocity", rclcpp::ParameterValue(30.0f));
    node_->get_parameter(name_ + "." + "init_max_velocity", params_.init_max_velocity);

    dogm_map_ = std::make_unique<dogm::DOGM>(params_);
    CHECK_ERROR(hipMalloc(&measurement_grid_, dogm_map_->grid_cell_count * sizeof(dogm::MeasurementCell)));

    robot_x_ = 0;
    robot_y_ = 0;
    is_first_measurement_ = true;

    // publisher_ = node_->create_publisher<dogm_msgs::msg::DynamicOccupancyGrid>("dogm_map", 10);
}

DogmLayer::~DogmLayer() {
    CHECK_ERROR(hipFree(measurement_grid_));
}

void DogmLayer::updateBounds(double robot_x, double robot_y, double robot_yaw,
                             double* min_x, double* min_y,
                             double* max_x, double* max_y) {
    *min_x = robot_x - dogm_map_->params.size / 2;
    *min_y = robot_y - dogm_map_->params.size / 2;
    *max_x = robot_x + dogm_map_->params.size / 2;
    *max_y = robot_y + dogm_map_->params.size / 2;
    robot_x_ = robot_x;
    robot_y_ = robot_y;
    return;
}

void DogmLayer::updateCosts(nav2_costmap_2d::Costmap2D& master_grid,
                            int min_i, int min_j, int max_i, int max_j) {
    if (!enabled_) {
        return;
    }

    auto time_stamp = node_->now();
    costMapToMeasurementGrid(master_grid, min_i, min_j, max_i, max_j, 0.5);
    float robot_x = 0.f;
    float robot_y = 0.f;
    if (motion_compensation_) {
        robot_x = robot_x_;
        robot_y = robot_y_;
    }
    if (!is_first_measurement_) {
        float dt = (time_stamp - last_time_stamp_).seconds();
        dogm_map_->updateGrid(measurement_grid_, robot_x, robot_y, 0, dt);
    } else {
        dogm_map_->updateGrid(measurement_grid_, robot_x, robot_y, 0, 0);
        is_first_measurement_ = false;
    }
    last_time_stamp_ = time_stamp;

    cv::Mat occupancy_image = dogm_map_->getOccupancyImage();
    int vis_image_size_ = 600;
    float vis_occupancy_threshold_ = 0.6;
    float vis_mahalanobis_distance_ = 2.0;
    dogm_map_->drawVelocities(occupancy_image, vis_image_size_, 1., vis_occupancy_threshold_, vis_mahalanobis_distance_);
    cv::namedWindow("occupancy_image", cv::WINDOW_NORMAL);
    cv::imshow("occupancy_image", occupancy_image);
    cv::waitKey(1);
}

void DogmLayer::costMapToMeasurementGrid(nav2_costmap_2d::Costmap2D& master_grid,
                                         int min_i, int min_j, int max_i, int max_j,
                                         float occupancy_threshold) {
    unsigned int size_x = master_grid.getSizeInCellsX();
    unsigned int size_y = master_grid.getSizeInCellsY();
    min_i = std::max(0, min_i);
    min_j = std::max(0, min_j);
    max_i = std::min(static_cast<int>(size_x), max_i);
    max_j = std::min(static_cast<int>(size_y), max_j);

    float measurement_grid_resolution = dogm_map_->params.resolution;
    float costmap_resolution = master_grid.getResolution();
    cv::Mat scale_measurement_grid(cv::Mat::eye(cv::Size(3, 3), CV_32F));
    float scale = costmap_resolution / measurement_grid_resolution;
    scale_measurement_grid.at<float>(0, 0) *= scale;
    scale_measurement_grid.at<float>(1, 1) *= scale;
    
    float measurement_grid_origin_x = robot_x_ - dogm_map_->params.size / 2;
    float measurement_grid_origin_y = robot_y_ - dogm_map_->params.size / 2;
    float costmap_origin_x = master_grid.getOriginX() + min_i * costmap_resolution;
    float costmap_origin_y = master_grid.getOriginY() + min_j * costmap_resolution;
    cv::Mat scaled_measurement_grid_to_costmap(cv::Mat::eye(cv::Size(3, 3), CV_32F));
    scaled_measurement_grid_to_costmap.at<float>(0, 2) = (costmap_origin_x - measurement_grid_origin_x) / costmap_resolution;
    scaled_measurement_grid_to_costmap.at<float>(1, 2) = (costmap_origin_y - measurement_grid_origin_y) / costmap_resolution;

    cv::Mat measurement_grid_to_costmap = scale_measurement_grid * scaled_measurement_grid_to_costmap;

    dim3 blocks(1, 1);
    dim3 threads(16, 16);
    unsigned char* master_array = master_grid.getCharMap();
    cv::Mat costmap(cv::Size(max_i - min_i, max_j - min_j), CV_8U, master_array + master_grid.getIndex(min_i, min_j), size_x * sizeof(unsigned char));
    costmap.convertTo(costmap, CV_32S);
    cv::cuda::GpuMat costmap_device;
    costmap_device.upload(costmap);
    setUnknownAsFree<<<blocks, threads>>>(costmap_device);

    cv::Mat measurement_grid;
    cv::cuda::GpuMat measurement_grid_device;
    cv::cuda::warpAffine(costmap_device, measurement_grid_device, measurement_grid_to_costmap(cv::Range(0, 2), cv::Range(0, 3)),
        cv::Size(dogm_map_->grid_size, dogm_map_->grid_size), cv::INTER_LINEAR, cv::BORDER_CONSTANT, cv::Scalar(nav2_costmap_2d::FREE_SPACE));
    fillMeasurementGrid<<<blocks, threads>>>(measurement_grid_, measurement_grid_device, occupancy_threshold);

    CHECK_ERROR(hipGetLastError());
    CHECK_ERROR(hipDeviceSynchronize());
}

void DogmLayer::publishDynamicGrid() {
    auto message = dogm_msgs::msg::DynamicOccupancyGrid();
    message.header.stamp = node_->now();
    message.header.frame_id = "abc";
    message.info.resolution = dogm_map_->getResolution();
    message.info.length = dogm_map_->getGridSize() * dogm_map_->getResolution();
    message.info.size = dogm_map_->getGridSize();
    message.info.pose.position.x = dogm_map_->getPositionX();
    message.info.pose.position.y = dogm_map_->getPositionY();
    message.info.pose.position.z = 0.0;
    message.info.pose.orientation.x = 0.0;
    message.info.pose.orientation.y = 0.0;
    message.info.pose.orientation.z = 0.0;
    message.info.pose.orientation.w = 1.0;

    message.data.clear();
    message.data.resize(dogm_map_->getGridSize() * dogm_map_->getGridSize());

    std::vector<dogm::GridCell> grid_cell_array = dogm_map_->getGridCells();
    #pragma omp parallel for
    for (int i = 0; i < message.data.size(); i++) {
        const dogm::GridCell& cell = grid_cell_array[i];

        message.data[i].free_mass = cell.free_mass;
        message.data[i].occ_mass = cell.occ_mass;

        message.data[i].mean_x_vel = cell.mean_x_vel;
        message.data[i].mean_y_vel = cell.mean_y_vel;
        message.data[i].var_x_vel = cell.var_x_vel;
        message.data[i].var_y_vel = cell.var_y_vel;
        message.data[i].covar_xy_vel = cell.covar_xy_vel;
    }

    publisher_->publish(message);
}

void DogmLayer::reset() {
    return;
}

__global__ void setUnknownAsFree(cv::cuda::PtrStepSzi occupancy_grid)
{
    int start_row = blockIdx.y * blockDim.y + threadIdx.y;
    int start_col = blockIdx.x * blockDim.x + threadIdx.x;
    int step_row = blockDim.y * gridDim.y;
    int step_col = blockDim.x * gridDim.x;
    for (int row = start_row; row < occupancy_grid.rows; row += step_row)
    {
        for (int col = start_col; col < occupancy_grid.cols; col += step_col)
        {
            if (occupancy_grid(row, col) == nav2_costmap_2d::NO_INFORMATION)
            {
                occupancy_grid(row, col) = nav2_costmap_2d::FREE_SPACE;
            }
        }
    }
}

__device__ float clip(float x, float min, float max)
{
    assert(min <= max);
    if (x < min) return min;
    if (x > max) return max;
    return x;
}

__global__ void fillMeasurementGrid(dogm::MeasurementCell* __restrict__ measurement_grid, const cv::cuda::PtrStepSzi source,
                                    float occupancy_threshold)
{
    int start_row = blockIdx.y * blockDim.y + threadIdx.y;
    int start_col = blockIdx.x * blockDim.x + threadIdx.x;
    int step_row = blockDim.y * gridDim.y;
    int step_col = blockDim.x * gridDim.x;
    const float eps = 0.0001f;
    for (int row = start_row; row < source.rows; row += step_row)
    {
        for (int col = start_col; col < source.cols; col += step_col)
        {
            int index = col + row * source.cols;
            float occ = 1.0f * source(row, col) / nav2_costmap_2d::LETHAL_OBSTACLE;
            if (occ < occupancy_threshold)
            {
                measurement_grid[index].free_mass = clip(1 - occ, eps, 1 - eps);
                measurement_grid[index].occ_mass = eps;
            }
            else
            {
                measurement_grid[index].free_mass = eps;
                measurement_grid[index].occ_mass = clip(occ, eps, 1 - eps);
            }
            measurement_grid[index].likelihood = 1.0f;
            measurement_grid[index].p_A = 1.0f;
        }
    }
}

}  // namespace dogm_plugin

#include "pluginlib/class_list_macros.hpp"
PLUGINLIB_EXPORT_CLASS(dogm_plugin::DogmLayer, nav2_costmap_2d::Layer)
